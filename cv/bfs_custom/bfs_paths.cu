#include <hip/hip_runtime.h>

#include <iostream>
#include <fstream>
#include <sstream>
#include <vector>
#include <queue>
#include <climits>
#include <algorithm>
#include <cassert>
#include <chrono>
#include <sys/resource.h>     // For getrusage() to measure peak CPU RAM usage
#include <filesystem>         // C++17 for std::filesystem::path, create_directories

// Uncomment for debug prints
// #define DEBUG_PRINT

// For GPU error-checking convenience
#define CUDA_CHECK(call)                                                         \
    do {                                                                         \
        hipError_t err = call;                                                 \
        if (err != hipSuccess) {                                               \
            std::cerr << "CUDA error at " << __FILE__ << ":" << __LINE__ << ": " \
                      << hipGetErrorString(err) << std::endl;                  \
            exit(EXIT_FAILURE);                                                 \
        }                                                                        \
    } while(0)

// -----------------------------------------------------------------------------
// Global variables to track peak GPU VRAM usage
// -----------------------------------------------------------------------------
static size_t g_totalGPUMem   = 0;      
static size_t g_minFreeMem    = SIZE_MAX; 
static bool   g_memInitialized = false;

void updateVRAMUsage() {
    hipDeviceSynchronize();
    size_t freeMem = 0, totalMem = 0;
    hipMemGetInfo(&freeMem, &totalMem);

    if (!g_memInitialized) {
        g_totalGPUMem   = totalMem;
        g_memInitialized = true;
    }
    if (freeMem < g_minFreeMem) {
        g_minFreeMem = freeMem;
    }
}

template <typename T>
inline void myCudaMalloc(T** ptr, size_t size) {
    CUDA_CHECK(hipMalloc((void**)ptr, size));
    updateVRAMUsage();
}

inline void myCudaFree(void* ptr) {
    CUDA_CHECK(hipFree(ptr));
    updateVRAMUsage();
}

// -----------------------------------------------------------------------------
// Graph Structure (adjacency list in CSR-like form)
// -----------------------------------------------------------------------------
struct Graph {
    int numVertices;
    int numEdges;
    std::vector<int> adjacencyList; 
    std::vector<int> edgesOffset;  
    std::vector<int> edgesSize;    
};

// -----------------------------------------------------------------------------
// Load the graph from an edge list file: each line "src dst"
// -----------------------------------------------------------------------------
void loadGraph(const std::string &filename, Graph &G)
{
    std::ifstream in(filename);
    if (!in.is_open()) {
        std::cerr << "Could not open file: " << filename << "\n";
        exit(EXIT_FAILURE);
    }

    std::vector<std::pair<int,int>> edges;
    int maxNodeId = -1;
    {
        std::string line;
        while (std::getline(in, line)) {
            if (line.empty()) continue;
            std::stringstream ss(line);
            int s, t;
            ss >> s >> t;
            edges.push_back({s, t});
            maxNodeId = std::max(maxNodeId, std::max(s, t));
        }
    }
    in.close();

    G.numVertices = maxNodeId + 1;
    G.numEdges    = static_cast<int>(edges.size());

    G.edgesOffset.resize(G.numVertices, 0);
    G.edgesSize.resize(G.numVertices,   0);

    // Count outdegree for each vertex
    for (auto &e : edges) {
        int s = e.first;
        G.edgesSize[s]++;
    }

    // Compute prefix sums for edgesOffset
    for (int i = 1; i < G.numVertices; i++) {
        G.edgesOffset[i] = G.edgesOffset[i-1] + G.edgesSize[i-1];
    }

    // Make a copy of offset as a "fill pointer"
    std::vector<int> fillPtr = G.edgesOffset;
    G.adjacencyList.resize(G.numEdges);

    // Fill adjacencyList
    for (auto &e : edges) {
        int s = e.first;
        int t = e.second;
        int pos = fillPtr[s]++;
        G.adjacencyList[pos] = t;
    }
}

// -----------------------------------------------------------------------------
// CUDA kernel: expand the front layer in parallel
// -----------------------------------------------------------------------------
__global__
void kernel_expand_front(int nFront,
                         const int *frontNodes,
                         const int *adjList,
                         const int *offsets,
                         const int *sizes,
                         int *outDegrees,
                         int *neighbors)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= nFront) return;

    int node = frontNodes[i];
    int start = offsets[node];
    int sz    = sizes[node];

    outDegrees[i] = sz;

    // Copy adjacency into the output neighbors array
    for (int j = 0; j < sz; j++) {
        neighbors[i + j * nFront] = adjList[start + j];
    }
}

// -----------------------------------------------------------------------------
// A custom compressed "visited" bitset
// -----------------------------------------------------------------------------
struct Bitset {
    std::vector<uint64_t> blocks;  // each block is 64 bits

    Bitset() = default;
    Bitset(int nVertices) {
        size_t nBlocks = (nVertices + 63) / 64;
        blocks.resize(nBlocks, 0ull);
    }
    
    // Copy constructor
    Bitset(const Bitset &other) = default;
    
    // Check if vertex v is set
    inline bool test(int v) const {
        int blockIdx = v / 64;
        int bitPos   = v % 64;
        return (blocks[blockIdx] & (1ULL << bitPos)) != 0ULL;
    }
    
    // Set vertex v
    inline void set(int v) {
        int blockIdx = v / 64;
        int bitPos   = v % 64;
        blocks[blockIdx] |= (1ULL << bitPos);
    }
};

// -----------------------------------------------------------------------------
// A "PathItem" that stores the partial path and a compressed visited bitset
// -----------------------------------------------------------------------------
struct PathItem {
    std::vector<int> path;  // The actual path of nodes so far
    Bitset visited;         // Which nodes are visited in path

    PathItem(int nVertices) : visited(nVertices) {}
    PathItem(const PathItem &other) = default; 
};

// -----------------------------------------------------------------------------
// BFS-like enumeration of all simple paths from start->end
// -----------------------------------------------------------------------------
void findAllPathsBFS_GPU(const Graph &G, int start, int end, 
                         std::vector<std::vector<int>> &allPaths)
{
    // Initial queue: a single path with [start]
    std::vector<PathItem> queue;
    {
        PathItem item(G.numVertices);
        item.path.push_back(start);
        item.visited.set(start);
        queue.push_back(std::move(item));
    }

    // Allocate adjacency on the GPU
    int *d_adjList = nullptr;
    int *d_offsets = nullptr;
    int *d_sizes   = nullptr;

    myCudaMalloc(&d_adjList, G.adjacencyList.size() * sizeof(int));
    myCudaMalloc(&d_offsets, G.numVertices         * sizeof(int));
    myCudaMalloc(&d_sizes,   G.numVertices         * sizeof(int));

    CUDA_CHECK(hipMemcpy(d_adjList, G.adjacencyList.data(),
                          G.adjacencyList.size()*sizeof(int),
                          hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_offsets, G.edgesOffset.data(),
                          G.numVertices*sizeof(int),
                          hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_sizes,   G.edgesSize.data(),
                          G.numVertices*sizeof(int),
                          hipMemcpyHostToDevice));

    // Expand BFS in "waves"
    while (!queue.empty()) {
        // 1) Gather the last node in each path
        int nFront = static_cast<int>(queue.size());
        std::vector<int> frontNodes(nFront);
        for (int i = 0; i < nFront; i++) {
            frontNodes[i] = queue[i].path.back();
        }

        // 2) Copy frontNodes to GPU
        int *d_frontNodes = nullptr;
        myCudaMalloc(&d_frontNodes, nFront * sizeof(int));
        CUDA_CHECK(hipMemcpy(d_frontNodes, frontNodes.data(),
                              nFront*sizeof(int), hipMemcpyHostToDevice));

        // 3) outDegrees array
        int *d_outDegrees = nullptr;
        myCudaMalloc(&d_outDegrees, nFront * sizeof(int));

        // 4) find maxOutDegree among these front nodes
        int maxOutDegree = 0;
        for (int fn : frontNodes) {
            maxOutDegree = std::max(maxOutDegree, G.edgesSize[fn]);
        }

        // 5) neighbors
        int *d_neighbors = nullptr;
        if (maxOutDegree > 0) {
            myCudaMalloc(&d_neighbors, nFront * maxOutDegree * sizeof(int));
        }

        // 6) Launch kernel
        {
            int blockSize = 128;
            int gridSize  = (nFront + blockSize - 1) / blockSize;
            kernel_expand_front<<<gridSize, blockSize>>>(
                nFront, d_frontNodes, d_adjList, d_offsets, d_sizes,
                d_outDegrees, d_neighbors
            );
            CUDA_CHECK(hipDeviceSynchronize());
        }

        // 7) Copy data back and build the next queue
        std::vector<int> outDegrees(nFront);
        std::vector<PathItem> nextQueue;
        if (maxOutDegree > 0) {
            std::vector<int> neighborsCPU(nFront * maxOutDegree);

            CUDA_CHECK(hipMemcpy(outDegrees.data(), d_outDegrees,
                                  nFront*sizeof(int), hipMemcpyDeviceToHost));
            CUDA_CHECK(hipMemcpy(neighborsCPU.data(), d_neighbors,
                                  nFront*maxOutDegree*sizeof(int), 
                                  hipMemcpyDeviceToHost));

            // For each partial path, expand with neighbors
            nextQueue.reserve(nFront * 2); // just a guess
            for (int i = 0; i < nFront; i++) {
                const PathItem &pitem = queue[i];
                int deg = outDegrees[i];
                for (int j = 0; j < deg; j++) {
                    int nbr = neighborsCPU[i + j*nFront];
                    if (pitem.visited.test(nbr)) {
                        // skip cycles
                        continue;
                    }
                    // build new path
                    PathItem newItem = pitem; // copy
                    newItem.path.push_back(nbr);
                    newItem.visited.set(nbr);

                    if (nbr == end) {
                        allPaths.push_back(newItem.path);
                    } else {
                        nextQueue.push_back(std::move(newItem));
                    }
                }
            }
        }

        queue.swap(nextQueue);

        // Cleanup
        myCudaFree(d_frontNodes);
        myCudaFree(d_outDegrees);
        if (maxOutDegree > 0) {
            myCudaFree(d_neighbors);
        }
    }

    // Free adjacency
    myCudaFree(d_adjList);
    myCudaFree(d_offsets);
    myCudaFree(d_sizes);
}

// -----------------------------------------------------------------------------
// Save all paths to JSON
// -----------------------------------------------------------------------------
void saveAllPaths(const std::vector<std::vector<int>> &allPaths,
                  const std::string &outFilename)
{
    std::ofstream out(outFilename);
    if (!out.is_open()) {
        std::cerr << "Cannot open output file: " << outFilename << "\n";
        return;
    }

    out << "[\n";
    for (size_t i = 0; i < allPaths.size(); i++) {
        out << "  [";
        for (size_t j = 0; j < allPaths[i].size(); j++) {
            out << allPaths[i][j];
            if (j + 1 < allPaths[i].size()) {
                out << ", ";
            }
        }
        out << "]";
        if (i + 1 < allPaths.size()) {
            out << ",";
        }
        out << "\n";
    }
    out << "]\n";

    out.close();
    std::cout << "Saved " << allPaths.size() 
              << " paths to " << outFilename << "\n";
}

// -----------------------------------------------------------------------------
// Main
// Usage: ./bfs_paths <edges.txt> <start> <end> <out_directory>
// -----------------------------------------------------------------------------
int main(int argc, char** argv)
{
    auto tStart = std::chrono::high_resolution_clock::now();

    if (argc < 5) {
        std::cerr << "Usage: " << argv[0]
                  << " <edges.txt> <start> <end> <out_directory>\n";
        return 1;
    }

    std::string filename   = argv[1];
    int         start      = std::stoi(argv[2]);
    int         end        = std::stoi(argv[3]);
    std::string outDirArg  = argv[4];

    // Ensure the output directory exists
    std::filesystem::path outDir(outDirArg);
    try {
        std::filesystem::create_directories(outDir);
    } catch (const std::exception &ex) {
        std::cerr << "Error creating output directory: " << ex.what() << "\n";
        return 1;
    }

    // Load graph
    Graph G;
    loadGraph(filename, G);

    std::cout << "Graph loaded with " << G.numVertices 
              << " vertices and " << G.numEdges << " edges.\n";

    if (start < 0 || start >= G.numVertices ||
        end   < 0 || end   >= G.numVertices)
    {
        std::cerr << "Start/end node out of range [0.."
                  << (G.numVertices - 1) << "]\n";
        return 2;
    }

    // Enumerate all paths
    std::vector<std::vector<int>> allPaths;
    findAllPathsBFS_GPU(G, start, end, allPaths);

    // Construct output filenames
    std::string baseFilename = std::filesystem::path(filename).stem().string();

    std::ostringstream outFile;
    outFile << "bfs_paths_" << start << "_" << end << "_" << baseFilename << ".json";
    std::filesystem::path outJsonPath = outDir / outFile.str();

    saveAllPaths(allPaths, outJsonPath.string());
    std::cout << "Total paths found: " << allPaths.size() << "\n";

    // Timing and resource usage
    auto tEnd = std::chrono::high_resolution_clock::now();
    double elapsedSec = std::chrono::duration<double>(tEnd - tStart).count();

    // Peak CPU RAM (KB, on Linux/Unix)
    struct rusage usage;
    getrusage(RUSAGE_SELF, &usage);
    long peakRamKB = usage.ru_maxrss;

    // Peak GPU usage
    size_t peakGpuBytes = 0;
    if (g_memInitialized) {
        peakGpuBytes = g_totalGPUMem - g_minFreeMem;
    }

    // Write stats
    std::ostringstream statsFilename;
    statsFilename << "bfs_stats_" << start << "_" << end << "_" << baseFilename << ".csv";
    std::filesystem::path outCsvPath = outDir / statsFilename.str();

    std::ofstream statsOut(outCsvPath);
    if (!statsOut.is_open()) {
        std::cerr << "Cannot open " << outCsvPath.string() << " for writing.\n";
    } else {
        statsOut << "Execution Time (seconds),Peak CPU RAM (MB),Peak GPU VRAM (MB)\n";
        double peakRamMB = static_cast<double>(peakRamKB) / 1024.0;
        double peakGpuMB = static_cast<double>(peakGpuBytes) / (1024.0 * 1024.0);
        statsOut << elapsedSec << "," 
                 << peakRamMB << "," 
                 << peakGpuMB << "\n";
        statsOut.close();
        std::cout << "Saved execution stats to " << outCsvPath.string() << "\n";
    }

    return 0;
}
