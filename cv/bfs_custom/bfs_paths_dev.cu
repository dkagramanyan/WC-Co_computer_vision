#include <hip/hip_runtime.h>

#include <iostream>
#include <fstream>
#include <sstream>
#include <vector>
#include <queue>
#include <climits>
#include <algorithm>
#include <cassert>
#include <chrono>
#include <sys/resource.h>     // For getrusage() to measure peak CPU RAM usage
#include <filesystem>         // C++17 for std::filesystem::path, create_directories

// Uncomment for debug prints
// #define DEBUG_PRINT

// For GPU error-checking convenience
#define CUDA_CHECK(call)                                                         \
    do {                                                                         \
        hipError_t err = call;                                                 \
        if (err != hipSuccess) {                                               \
            std::cerr << "CUDA error at " << __FILE__ << ":" << __LINE__ << ": " \
                      << hipGetErrorString(err) << std::endl;                  \
            exit(EXIT_FAILURE);                                                 \
        }                                                                        \
    } while(0)

// -----------------------------------------------------------------------------
// Global variables to track peak GPU VRAM usage
// -----------------------------------------------------------------------------
static size_t g_totalGPUMem   = 0;      
static size_t g_minFreeMem    = SIZE_MAX; 
static bool   g_memInitialized = false;

void updateVRAMUsage() {
    hipDeviceSynchronize();
    size_t freeMem = 0, totalMem = 0;
    hipMemGetInfo(&freeMem, &totalMem);

    if (!g_memInitialized) {
        g_totalGPUMem   = totalMem;
        g_memInitialized = true;
    }
    if (freeMem < g_minFreeMem) {
        g_minFreeMem = freeMem;
    }
}

template <typename T>
inline void myCudaMalloc(T** ptr, size_t size) {
    CUDA_CHECK(hipMalloc((void**)ptr, size));
    updateVRAMUsage();
}

inline void myCudaFree(void* ptr) {
    CUDA_CHECK(hipFree(ptr));
    updateVRAMUsage();
}

// -----------------------------------------------------------------------------
// Graph Structure (adjacency list in CSR-like form)
// -----------------------------------------------------------------------------
struct Graph {
    int numVertices;
    int numEdges;
    std::vector<int> adjacencyList; 
    std::vector<int> edgesOffset;  
    std::vector<int> edgesSize;    
};

// -----------------------------------------------------------------------------
// Load the graph from an edge list file: each line "src dst"
// -----------------------------------------------------------------------------
void loadGraph(const std::string &filename, Graph &G)
{
    std::ifstream in(filename);
    if (!in.is_open()) {
        std::cerr << "Could not open file: " << filename << "\n";
        exit(EXIT_FAILURE);
    }

    std::vector<std::pair<int,int>> edges;
    int maxNodeId = -1;
    {
        std::string line;
        while (std::getline(in, line)) {
            if (line.empty()) continue;
            std::stringstream ss(line);
            int s, t;
            ss >> s >> t;
            edges.push_back({s, t});
            maxNodeId = std::max(maxNodeId, std::max(s, t));
        }
    }
    in.close();

    G.numVertices = maxNodeId + 1;
    G.numEdges    = static_cast<int>(edges.size());

    G.edgesOffset.resize(G.numVertices, 0);
    G.edgesSize.resize(G.numVertices,   0);

    // Count outdegree for each vertex
    for (auto &e : edges) {
        int s = e.first;
        G.edgesSize[s]++;
    }

    // Compute prefix sums for edgesOffset
    for (int i = 1; i < G.numVertices; i++) {
        G.edgesOffset[i] = G.edgesOffset[i-1] + G.edgesSize[i-1];
    }

    // Make a copy of offset as a "fill pointer"
    std::vector<int> fillPtr = G.edgesOffset;
    G.adjacencyList.resize(G.numEdges);

    // Fill adjacencyList
    for (auto &e : edges) {
        int s = e.first;
        int t = e.second;
        int pos = fillPtr[s]++;
        G.adjacencyList[pos] = t;
    }
}

// -----------------------------------------------------------------------------
// CUDA kernel: expand the front layer in parallel
// -----------------------------------------------------------------------------
__global__
void kernel_expand_front(int nFront,
                         const int *frontNodes,
                         const int *adjList,
                         const int *offsets,
                         const int *sizes,
                         int *outDegrees,
                         int *neighbors)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= nFront) return;

    int node = frontNodes[i];
    int start = offsets[node];
    int sz    = sizes[node];

    outDegrees[i] = sz;

    // Copy adjacency into the output neighbors array
    for (int j = 0; j < sz; j++) {
        neighbors[i + j * nFront] = adjList[start + j];
    }
}

// -----------------------------------------------------------------------------
// A custom compressed "visited" bitset
// -----------------------------------------------------------------------------
struct Bitset {
    std::vector<uint64_t> blocks;  // each block is 64 bits

    Bitset() = default;
    explicit Bitset(int nVertices) {
        size_t nBlocks = (nVertices + 63) / 64;
        blocks.resize(nBlocks, 0ull);
    }
    
    // Check if vertex v is set
    inline bool test(int v) const {
        int blockIdx = v / 64;
        int bitPos   = v % 64;
        return (blocks[blockIdx] & (1ULL << bitPos)) != 0ULL;
    }
    
    // Set vertex v
    inline void set(int v) {
        int blockIdx = v / 64;
        int bitPos   = v % 64;
        blocks[blockIdx] |= (1ULL << bitPos);
    }
};

// -----------------------------------------------------------------------------
// A "PathItem" that stores the partial path and a compressed visited bitset
// -----------------------------------------------------------------------------
struct PathItem {
    // The partial path of nodes
    std::vector<int> path;  
    // Which nodes are visited in path
    Bitset visited;         

    PathItem() = default;
    PathItem(int nVertices) : visited(nVertices) {}
};

// -----------------------------------------------------------------------------
// Write a PathItem to disk in a binary format
// -----------------------------------------------------------------------------
void writePathItem(std::ofstream &out, const PathItem &item) {
    // 1) Write path size + path data
    size_t pathLen = item.path.size();
    out.write(reinterpret_cast<const char*>(&pathLen), sizeof(pathLen));
    out.write(reinterpret_cast<const char*>(item.path.data()),
              pathLen * sizeof(int));
    // 2) Write visited bitset blocks
    size_t nBlocks = item.visited.blocks.size();
    out.write(reinterpret_cast<const char*>(&nBlocks), sizeof(nBlocks));
    out.write(reinterpret_cast<const char*>(item.visited.blocks.data()),
              nBlocks * sizeof(uint64_t));
}

// -----------------------------------------------------------------------------
// Read a PathItem from disk
// -----------------------------------------------------------------------------
void readPathItem(std::ifstream &in, PathItem &item) {
    // 1) Read path length + path
    size_t pathLen;
    in.read(reinterpret_cast<char*>(&pathLen), sizeof(pathLen));
    item.path.resize(pathLen);
    in.read(reinterpret_cast<char*>(item.path.data()), pathLen * sizeof(int));

    // 2) Read visited bitset
    size_t nBlocks = 0;
    in.read(reinterpret_cast<char*>(&nBlocks), sizeof(nBlocks));
    item.visited.blocks.resize(nBlocks, 0ULL);
    in.read(reinterpret_cast<char*>(item.visited.blocks.data()),
            nBlocks * sizeof(uint64_t));
}

// -----------------------------------------------------------------------------
// Flush all items to disk (overwrites the file)
// -----------------------------------------------------------------------------
void flushToDisk(const std::vector<PathItem> &items, const std::string &filename) {
    std::ofstream out(filename, std::ios::binary | std::ios::trunc);
    if (!out.is_open()) {
        std::cerr << "Cannot open cache file " << filename << " for writing.\n";
        return;
    }

    // Write how many items
    size_t count = items.size();
    out.write(reinterpret_cast<const char*>(&count), sizeof(count));

    // Write each PathItem
    for (const auto &it : items) {
        writePathItem(out, it);
    }
    out.close();
}

// -----------------------------------------------------------------------------
// Read all PathItem from disk
// -----------------------------------------------------------------------------
void loadFromDisk(std::vector<PathItem> &items, const std::string &filename) {
    std::ifstream in(filename, std::ios::binary);
    if (!in.is_open()) {
        std::cerr << "Cannot open cache file " << filename << " for reading.\n";
        return;
    }

    // Read how many
    size_t count = 0;
    in.read(reinterpret_cast<char*>(&count), sizeof(count));

    items.clear();
    items.reserve(count);
    for (size_t i = 0; i < count; i++) {
        PathItem temp;
        readPathItem(in, temp);
        items.push_back(std::move(temp));
    }
    in.close();
}

// -----------------------------------------------------------------------------
// BFS-like enumeration of all simple paths from start->end
// with disk caching each BFS wave
// -----------------------------------------------------------------------------
void findAllPathsBFS_GPU(const Graph &G, int start, int end, 
                         std::vector<std::vector<int>> &allPaths,
                         const std::string &outDir)
{
    // We'll store expansions for each BFS wave in "wave_cache.bin" under outDir
    std::filesystem::path cacheFilePath = 
        std::filesystem::path(outDir) / "wave_cache.bin";

    // Prepare initial wave with a single path
    std::vector<PathItem> queue;
    {
        PathItem item(G.numVertices);
        item.path.push_back(start);
        item.visited.set(start);
        queue.push_back(std::move(item));
    }

    // GPU adjacency
    int *d_adjList   = nullptr;
    int *d_offsets   = nullptr;
    int *d_sizes     = nullptr;

    myCudaMalloc(&d_adjList, G.adjacencyList.size() * sizeof(int));
    myCudaMalloc(&d_offsets, G.numVertices         * sizeof(int));
    myCudaMalloc(&d_sizes,   G.numVertices         * sizeof(int));

    CUDA_CHECK(hipMemcpy(d_adjList, G.adjacencyList.data(),
                          G.adjacencyList.size()*sizeof(int),
                          hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_offsets, G.edgesOffset.data(),
                          G.numVertices*sizeof(int),
                          hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_sizes,   G.edgesSize.data(),
                          G.numVertices*sizeof(int),
                          hipMemcpyHostToDevice));

    // BFS expansions in "waves"
    // Each iteration expands the 'queue' into 'nextQueue', then flushes nextQueue to disk.
    // Then we read it back from disk to become the next wave's queue.

    int waveIdx = 0;
    while (!queue.empty()) {
        waveIdx++;

        // 1) Gather last nodes
        int nFront = static_cast<int>(queue.size());
        std::vector<int> frontNodes(nFront);
        for (int i = 0; i < nFront; i++) {
            frontNodes[i] = queue[i].path.back();
        }

        // 2) Copy frontNodes to GPU
        int *d_frontNodes = nullptr;
        myCudaMalloc(&d_frontNodes, nFront * sizeof(int));
        CUDA_CHECK(hipMemcpy(d_frontNodes, frontNodes.data(),
                              nFront*sizeof(int), hipMemcpyHostToDevice));

        // 3) outDegrees array
        int *d_outDegrees = nullptr;
        myCudaMalloc(&d_outDegrees, nFront * sizeof(int));

        // 4) find maxOutDegree among these front nodes
        int maxOutDegree = 0;
        for (int fn : frontNodes) {
            maxOutDegree = std::max(maxOutDegree, G.edgesSize[fn]);
        }

        // 5) neighbors
        int *d_neighbors = nullptr;
        if (maxOutDegree > 0) {
            myCudaMalloc(&d_neighbors, nFront * maxOutDegree * sizeof(int));
        }

        // 6) GPU kernel
        {
            int blockSize = 128;
            int gridSize  = (nFront + blockSize - 1) / blockSize;
            kernel_expand_front<<<gridSize, blockSize>>>(
                nFront, d_frontNodes, d_adjList, d_offsets, d_sizes,
                d_outDegrees, d_neighbors
            );
            CUDA_CHECK(hipDeviceSynchronize());
        }

        // 7) Copy back, build nextWave
        std::vector<int> outDegrees(nFront);
        std::vector<PathItem> nextWave; 
        // We'll store expansions from this wave in nextWave, then flush it to disk

        if (maxOutDegree > 0) {
            std::vector<int> neighborsCPU(nFront * maxOutDegree);

            CUDA_CHECK(hipMemcpy(outDegrees.data(), d_outDegrees,
                                  nFront*sizeof(int), hipMemcpyDeviceToHost));
            CUDA_CHECK(hipMemcpy(neighborsCPU.data(), d_neighbors,
                                  nFront*maxOutDegree*sizeof(int), 
                                  hipMemcpyDeviceToHost));

            nextWave.reserve(nFront * 2); 
            for (int i = 0; i < nFront; i++) {
                const PathItem &pitem = queue[i];
                int deg = outDegrees[i];
                for (int j = 0; j < deg; j++) {
                    int nbr = neighborsCPU[i + j*nFront];
                    if (pitem.visited.test(nbr)) {
                        // skip cycles
                        continue;
                    }
                    // build new path
                    PathItem newItem = pitem; // copy
                    newItem.path.push_back(nbr);
                    newItem.visited.set(nbr);

                    if (nbr == end) {
                        allPaths.push_back(newItem.path);
                    } else {
                        nextWave.push_back(std::move(newItem));
                    }
                }
            }
        }

        // Cleanup for this wave
        myCudaFree(d_frontNodes);
        myCudaFree(d_outDegrees);
        if (maxOutDegree > 0) {
            myCudaFree(d_neighbors);
        }

        // 8) Flush nextWave to disk, clear it from RAM
        flushToDisk(nextWave, cacheFilePath.string());
        nextWave.clear();

        // 9) Now we've fully expanded the current queue -> nextWave on disk.
        //    Clear the old queue from RAM:
        queue.clear();

        // 10) Read from disk to load the expansions as the next BFS wave
        loadFromDisk(queue, cacheFilePath.string());

        // optionally remove the cache file (it'll be overwritten each wave)
        // std::filesystem::remove(cacheFilePath);
        // But it's fine to leave it for debug
    }

    // Free adjacency
    myCudaFree(d_adjList);
    myCudaFree(d_offsets);
    myCudaFree(d_sizes);
}

// -----------------------------------------------------------------------------
// Save all final paths to JSON
// -----------------------------------------------------------------------------
void saveAllPaths(const std::vector<std::vector<int>> &allPaths,
                  const std::string &outFilename)
{
    std::ofstream out(outFilename);
    if (!out.is_open()) {
        std::cerr << "Cannot open output file: " << outFilename << "\n";
        return;
    }

    out << "[\n";
    for (size_t i = 0; i < allPaths.size(); i++) {
        out << "  [";
        for (size_t j = 0; j < allPaths[i].size(); j++) {
            out << allPaths[i][j];
            if (j + 1 < allPaths[i].size()) {
                out << ", ";
            }
        }
        out << "]";
        if (i + 1 < allPaths.size()) {
            out << ",";
        }
        out << "\n";
    }
    out << "]\n";
    out.close();
    std::cout << "Saved " << allPaths.size() 
              << " paths to " << outFilename << "\n";
}

// -----------------------------------------------------------------------------
// Main
// Usage: ./bfs_paths <edges.txt> <start> <end> <out_directory>
// -----------------------------------------------------------------------------
int main(int argc, char** argv)
{
    auto tStart = std::chrono::high_resolution_clock::now();

    if (argc < 5) {
        std::cerr << "Usage: " << argv[0]
                  << " <edges.txt> <start> <end> <out_directory>\n";
        return 1;
    }

    std::string filename   = argv[1];
    int         start      = std::stoi(argv[2]);
    int         end        = std::stoi(argv[3]);
    std::string outDirArg  = argv[4];

    // Ensure the output directory exists
    std::filesystem::path outDir(outDirArg);
    try {
        std::filesystem::create_directories(outDir);
    } catch (const std::exception &ex) {
        std::cerr << "Error creating output directory: " << ex.what() << "\n";
        return 1;
    }

    // Load graph
    Graph G;
    loadGraph(filename, G);

    std::cout << "Graph loaded with " << G.numVertices 
              << " vertices and " << G.numEdges << " edges.\n";

    if (start < 0 || start >= G.numVertices ||
        end   < 0 || end   >= G.numVertices)
    {
        std::cerr << "Start/end node out of range [0.."
                  << (G.numVertices - 1) << "]\n";
        return 2;
    }

    // Enumerate all paths, using BFS + disk-caching
    std::vector<std::vector<int>> allPaths;
    findAllPathsBFS_GPU(G, start, end, allPaths, outDirArg);

    // Construct output filenames
    std::string baseFilename = std::filesystem::path(filename).stem().string();

    std::ostringstream outFile;
    outFile << "bfs_paths_" << start << "_" << end << "_" << baseFilename << ".json";
    std::filesystem::path outJsonPath = outDir / outFile.str();

    // Save final paths
    saveAllPaths(allPaths, outJsonPath.string());
    std::cout << "Total paths found: " << allPaths.size() << "\n";

    // Timing and resource usage
    auto tEnd = std::chrono::high_resolution_clock::now();
    double elapsedSec = std::chrono::duration<double>(tEnd - tStart).count();

    // Peak CPU RAM (KB, on Linux/Unix)
    struct rusage usage;
    getrusage(RUSAGE_SELF, &usage);
    long peakRamKB = usage.ru_maxrss;

    // Peak GPU usage
    size_t peakGpuBytes = 0;
    if (g_memInitialized) {
        peakGpuBytes = g_totalGPUMem - g_minFreeMem;
    }

    // Write stats
    std::ostringstream statsFilename;
    statsFilename << "bfs_stats_" << start << "_" << end << "_" << baseFilename << ".csv";
    std::filesystem::path outCsvPath = outDir / statsFilename.str();

    std::ofstream statsOut(outCsvPath);
    if (!statsOut.is_open()) {
        std::cerr << "Cannot open " << outCsvPath.string() << " for writing.\n";
    } else {
        statsOut << "Execution Time (seconds),Peak CPU RAM (MB),Peak GPU VRAM (MB)\n";
        double peakRamMB = static_cast<double>(peakRamKB) / 1024.0;
        double peakGpuMB = static_cast<double>(peakGpuBytes) / (1024.0 * 1024.0);
        statsOut << elapsedSec << "," 
                 << peakRamMB << "," 
                 << peakGpuMB << "\n";
        statsOut.close();
        std::cout << "Saved execution stats to " << outCsvPath.string() << "\n";
    }

    return 0;
}
