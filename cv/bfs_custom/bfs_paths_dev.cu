#include <hip/hip_runtime.h>

#include <iostream>
#include <fstream>
#include <sstream>
#include <vector>
#include <queue>
#include <climits>
#include <algorithm>
#include <cassert>
#include <chrono>
#include <sys/resource.h>  // For getrusage() to measure peak CPU RAM usage
#include <filesystem> // C++17 for std::filesystem::path

// Uncomment for debug prints
// #define DEBUG_PRINT

// For GPU error-checking convenience
#define CUDA_CHECK(call)                                                         \
    do {                                                                         \
        hipError_t err = call;                                                 \
        if (err != hipSuccess) {                                               \
            std::cerr << "CUDA error at " << __FILE__ << ":" << __LINE__ << ": " \
                      << hipGetErrorString(err) << std::endl;                  \
            exit(EXIT_FAILURE);                                                 \
        }                                                                        \
    } while(0)

// -----------------------------------------------------------------------------
// Global variables to track peak GPU VRAM usage
// -----------------------------------------------------------------------------
static size_t g_totalGPUMem = 0;         // total GPU memory (constant per device)
static size_t g_minFreeMem = SIZE_MAX;   // smallest "free memory" encountered
static bool   g_memInitialized = false;

// A small utility function to update the global minimum free GPU memory
// after each allocation/free. The difference (total - g_minFreeMem) is
// effectively the maximum usage we've seen.
void updateVRAMUsage() {
    // Ensure all operations have completed
    hipDeviceSynchronize();
    // Query current free/total memory
    size_t freeMem = 0, totalMem = 0;
    hipMemGetInfo(&freeMem, &totalMem);
    // Initialize our global total memory (once only)
    if (!g_memInitialized) {
        g_totalGPUMem = totalMem;
        g_memInitialized = true;
    }
    // Track the minimum free memory
    if (freeMem < g_minFreeMem) {
        g_minFreeMem = freeMem;
    }
}

// We define custom wrappers for cudaMalloc/cudaFree
// so that each time we allocate/free, we update usage info.
template <typename T>
inline void myCudaMalloc(T** ptr, size_t size) {
    CUDA_CHECK(hipMalloc((void**)ptr, size));
    updateVRAMUsage();
}

inline void myCudaFree(void* ptr) {
    CUDA_CHECK(hipFree(ptr));
    updateVRAMUsage();
}

// -----------------------------------------------------------------------------
// Graph Structure (adjacency list in CSR-like form)
// -----------------------------------------------------------------------------
struct Graph {
    int numVertices;
    int numEdges;
    std::vector<int> adjacencyList; // All edges flattened
    std::vector<int> edgesOffset;   // Where each vertex's adjacency starts
    std::vector<int> edgesSize;     // How many neighbors each vertex has
};

// -----------------------------------------------------------------------------
// Load the graph from an edge list file: each line "src dst"
// -----------------------------------------------------------------------------
void loadGraph(const std::string &filename, Graph &G)
{
    std::ifstream in(filename);
    if (!in.is_open()) {
        std::cerr << "Could not open file: " << filename << "\n";
        exit(EXIT_FAILURE);
    }

    std::vector<std::pair<int,int>> edges;
    int maxNodeId = -1;
    {
        std::string line;
        while (std::getline(in, line)) {
            if (line.empty()) continue;
            std::stringstream ss(line);
            int s, t;
            ss >> s >> t;
            edges.push_back({s, t});
            maxNodeId = std::max(maxNodeId, std::max(s, t));
        }
    }
    in.close();

    G.numVertices = maxNodeId + 1;
    G.numEdges    = (int)edges.size();

    G.edgesOffset.resize(G.numVertices, 0);
    G.edgesSize  .resize(G.numVertices, 0);

    // Count outdegree for each vertex
    for (auto &e : edges) {
        int s = e.first;
        G.edgesSize[s]++;
    }

    // Compute prefix sums for edgesOffset
    for (int i = 1; i < G.numVertices; i++) {
        G.edgesOffset[i] = G.edgesOffset[i-1] + G.edgesSize[i-1];
    }

    // Make a copy of offset as a "fill pointer"
    std::vector<int> fillPtr = G.edgesOffset;
    G.adjacencyList.resize(G.numEdges);

    // Fill adjacencyList
    for (auto &e : edges) {
        int s = e.first;
        int t = e.second;
        int pos = fillPtr[s]++;
        G.adjacencyList[pos] = t;
    }
}

// -----------------------------------------------------------------------------
// CUDA kernel: given a set of vertices in frontNodes, look up their neighbors
// in parallel. We'll collect all neighbors in a big array. We also store
// how many neighbors each vertex has in outDegrees[i], so that the CPU can know
// how to slice the neighbor array correctly later.
// -----------------------------------------------------------------------------
__global__
void kernel_expand_front(int nFront,
                         const int *frontNodes,
                         const int *adjList,
                         const int *offsets,
                         const int *sizes,
                         int *outDegrees,
                         int *neighbors)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= nFront) return;

    int node = frontNodes[i];
    int start = offsets[node];
    int sz    = sizes[node];

    // Write the out-degree
    outDegrees[i] = sz;

    // Copy adjacency into the output neighbors array
    for (int j = 0; j < sz; j++) {
        neighbors[i + j * nFront] = adjList[start + j];
    }
}

// -----------------------------------------------------------------------------
// A "PathItem" holds:
//   - path: the sequence of visited nodes (so far)
//   - visited: a boolean array to mark which nodes were already visited
//
// We do BFS in "waves" where each wave is a collection of PathItem's.
// -----------------------------------------------------------------------------
struct PathItem {
    std::vector<int> path;
    std::vector<bool> visited;  // visited[v] = true if 'v' is already in path

    PathItem(int n) : visited(n, false) {}
};

// -----------------------------------------------------------------------------
// BFS-like enumeration of all simple paths from start->end using adjacency
// expansions on the GPU. We store all partial paths in a queue on the CPU.
//
// If your graph has cycles or is large, this can blow up in memory/time!!
// -----------------------------------------------------------------------------
void findAllPathsBFS_GPU(const Graph &G, int start, int end, 
                         std::vector<std::vector<int>> &allPaths)
{
    // Initial queue: one path containing [start]
    std::vector<PathItem> queue;
    {
        PathItem item(G.numVertices);
        item.path.push_back(start);
        item.visited[start] = true;
        queue.push_back(std::move(item));
    }

    // Allocate GPU adjacency
    int *d_adjList   = nullptr;
    int *d_offsets   = nullptr;
    int *d_sizes     = nullptr;

    // Use our custom wrappers to track memory usage
    myCudaMalloc(&d_adjList,   G.adjacencyList.size() * sizeof(int));
    myCudaMalloc(&d_offsets,   G.numVertices         * sizeof(int));
    myCudaMalloc(&d_sizes,     G.numVertices         * sizeof(int));

    CUDA_CHECK(hipMemcpy(d_adjList, G.adjacencyList.data(),
                          G.adjacencyList.size()*sizeof(int), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_offsets, G.edgesOffset.data(),
                          G.numVertices*sizeof(int), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_sizes,   G.edgesSize.data(),
                          G.numVertices*sizeof(int), hipMemcpyHostToDevice));

    // We'll expand BFS in "waves" until no more expansions are possible.
    // But we store *all* partial paths that haven't yet reached 'end'.
    while (true) {
        if (queue.empty()) {
            break; // no more partial paths to expand
        }

        // 1) Collect the last node of each path in queue[] into frontNodes[]
        const int nFront = (int)queue.size();
        std::vector<int> frontNodes(nFront);
        for (int i = 0; i < nFront; i++) {
            frontNodes[i] = queue[i].path.back(); 
        }

        // 2) Copy frontNodes[] to GPU
        int *d_frontNodes = nullptr;
        myCudaMalloc(&d_frontNodes, nFront * sizeof(int));
        CUDA_CHECK(hipMemcpy(d_frontNodes, frontNodes.data(),
                              nFront*sizeof(int), hipMemcpyHostToDevice));

        // 3) For each of the nFront nodes, we have edgesSize[node] neighbors.
        //    We want to store the total neighbors in outDegrees[] so we can
        //    figure out how to slice them on the CPU side.
        int *d_outDegrees = nullptr;
        myCudaMalloc(&d_outDegrees, nFront * sizeof(int));

        // 4) We'll store neighbors in a 2D layout [i + j*nFront].
        //    First find the maxOutDegree among these frontNodes.
        int maxOutDegree = 0;
        for (int fn : frontNodes) {
            maxOutDegree = std::max(maxOutDegree, G.edgesSize[fn]);
        }

        // 5) Allocate device array for neighbors
        int *d_neighbors = nullptr;
        if (maxOutDegree > 0) {
            myCudaMalloc(&d_neighbors, nFront * maxOutDegree * sizeof(int));
        }

        // 6) Launch kernel to fill outDegrees[] and neighbors[]
        {
            int blockSize = 128;
            int gridSize  = (nFront + blockSize - 1) / blockSize;
            kernel_expand_front<<<gridSize, blockSize>>>(
                nFront, d_frontNodes, d_adjList, d_offsets, d_sizes,
                d_outDegrees, d_neighbors
            );
            CUDA_CHECK(hipDeviceSynchronize());
        }

        // 7) Copy outDegrees + neighbors back to CPU
        std::vector<int> outDegrees(nFront);
        if (maxOutDegree > 0) {
            std::vector<int> neighborsCPU(nFront * maxOutDegree);
            CUDA_CHECK(hipMemcpy(outDegrees.data(), d_outDegrees, 
                                  nFront*sizeof(int), hipMemcpyDeviceToHost));
            CUDA_CHECK(hipMemcpy(neighborsCPU.data(), d_neighbors,
                                  nFront*maxOutDegree*sizeof(int), hipMemcpyDeviceToHost));

            // 8) Build the next wave of partial paths by expanding
            //    each path in the queue with all its neighbors
            std::vector<PathItem> nextQueue;
            nextQueue.reserve(nFront * 2); // rough guess

            for (int i = 0; i < nFront; i++) {
                const PathItem &pitem = queue[i];
                int deg = outDegrees[i];
                for (int j = 0; j < deg; j++) {
                    int nbr = neighborsCPU[i + j*nFront];
                    // skip if already visited
                    if (pitem.visited[nbr]) {
                        continue;
                    }
                    // create a new path
                    PathItem newItem = pitem; // copy
                    newItem.path.push_back(nbr);
                    newItem.visited[nbr] = true;

                    // if we reached 'end', store it
                    if (nbr == end) {
                        allPaths.push_back(newItem.path);
                    } else {
                        nextQueue.push_back(std::move(newItem));
                    }
                }
            }
            queue.swap(nextQueue);
        }
        else {
            queue.clear();
        }

        // Clean up this layer
        myCudaFree(d_frontNodes);
        myCudaFree(d_outDegrees);
        if (maxOutDegree > 0) {
            myCudaFree(d_neighbors);
        }
    }

    // Clean up adjacency on device
    myCudaFree(d_adjList);
    myCudaFree(d_offsets);
    myCudaFree(d_sizes);
}

// -----------------------------------------------------------------------------
// Save all paths to a text file
// -----------------------------------------------------------------------------
void saveAllPaths(const std::vector<std::vector<int>> &allPaths,
                  const std::string &outFilename)
{
    std::ofstream out(outFilename);
    if (!out.is_open()) {
        std::cerr << "Cannot open output file: " << outFilename << "\n";
        return;
    }
    for (size_t i = 0; i < allPaths.size(); i++) {
        for (size_t j = 0; j < allPaths[i].size(); j++) {
            out << allPaths[i][j];
            if (j+1 < allPaths[i].size()) {
                out << " -> ";
            }
        }
        out << "\n";
    }
    out.close();
    std::cout << "Saved " << allPaths.size() << " paths to " << outFilename << "\n";
}

// -----------------------------------------------------------------------------
// Main
// Usage: ./bfs_paths edges.txt start end
// -----------------------------------------------------------------------------
int main(int argc, char** argv)
{
    // -------------------------------------------------------------------------
    // 1) Start timing
    // -------------------------------------------------------------------------
    auto tStart = std::chrono::high_resolution_clock::now();

    if (argc < 4) {
        std::cerr << "Usage: " << argv[0] << " edges.txt start end\n";
        return 1;
    }
    std::string filename = argv[1];
    int start = std::stoi(argv[2]);
    int end   = std::stoi(argv[3]);

    // -------------------------------------------------------------------------
    // 2) Load graph
    // -------------------------------------------------------------------------
    Graph G;
    loadGraph(filename, G);

    std::cout << "Graph loaded with " << G.numVertices 
              << " vertices and " << G.numEdges << " edges.\n";

    if (start < 0 || start >= G.numVertices ||
        end   < 0 || end   >= G.numVertices)
    {
        std::cerr << "Start or end node out of range [0.."
                  << (G.numVertices - 1) << "]\n";
        return 2;
    }

    // -------------------------------------------------------------------------
    // 3) Enumerate *all possible simple paths* from start -> end
    // -------------------------------------------------------------------------
    std::vector<std::vector<int>> allPaths;
    findAllPathsBFS_GPU(G, start, end, allPaths);

    // -------------------------------------------------------------------------
    // 4) Save results to a text file
    // -------------------------------------------------------------------------
    std::ostringstream outFile;
    std::string baseFilename = std::filesystem::path(filename).stem().string();
    outFile << "bfs_paths_" << start << "_" << end <<"_" << baseFilename << ".txt";

    saveAllPaths(allPaths, outFile.str());
    std::cout << "Total paths found: " << allPaths.size() << "\n";

    // -------------------------------------------------------------------------
    // 5) Stop timing + get CPU peak RAM usage
    // -------------------------------------------------------------------------
    auto tEnd = std::chrono::high_resolution_clock::now();
    double elapsedSec = std::chrono::duration<double>(tEnd - tStart).count();

    struct rusage usage;
    getrusage(RUSAGE_SELF, &usage);
    // Peak resident set size in kilobytes on Linux/Unix
    long peakRamKB = usage.ru_maxrss;

    // -------------------------------------------------------------------------
    // 6) Compute peak GPU usage
    // -------------------------------------------------------------------------
    // g_minFreeMem is the minimum "free memory" seen
    // g_totalGPUMem is the total GPU memory we recorded
    // So peak usage is (total - minFree).
    size_t peakGpuBytes = 0;
    if (g_memInitialized) {
        peakGpuBytes = g_totalGPUMem - g_minFreeMem;
    }

    // -------------------------------------------------------------------------
    // 7) Write statistics to a file
    // -------------------------------------------------------------------------
    {   
        std::ostringstream stats_filename;
        stats_filename << "bfs_stats_" << start << "_" << end << "_" << baseFilename << ".csv"; 
    
        std::ofstream statsOut(stats_filename.str());
        if (!statsOut.is_open()) {
            std::cerr << "Cannot open " << stats_filename.str() << " for writing.\n";
        } else {
            // Write CSV headers
            statsOut << "Execution Time (seconds),Peak CPU RAM (MB),Peak GPU VRAM (MB)\n";
            
            // Write data values
            statsOut << elapsedSec << "," 
                     << (peakRamKB / 1024) << "," 
                     << (peakGpuBytes / (1024 * 1024)) << "\n";
            
            statsOut.close();
            std::cout << "Saved execution stats to " << stats_filename.str() << "\n";
        }
    }


    return 0;
}
